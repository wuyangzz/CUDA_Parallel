#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include <time.h>

#define MAX_SIZE 1024
#define M_SIZE MAX_SIZE * MAX_SIZE 
#define GRID_SIZE  1024    //ÿ��grid����1024��block
#define BLOCK_SIZE  1024 //1070���Ϊÿ��block����1024���߳�

__global__ void MatrixMutiplyG(float* matrixM, float* matrixN, float* matrixC) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;

    float sum = 0, m, n;
    for (int k = 0; k < MAX_SIZE; ++k) {
        m = matrixM[bx * MAX_SIZE + k];
        n = matrixN[k * MAX_SIZE + tx];
        sum += m * n;
    }
    matrixC[bx * MAX_SIZE + tx] = sum;
}

int main()
{

    clock_t start, finish;
    float* matrixM, * matrixN, * matrixC;
    float* g_matrixM, * g_matrixN, * g_matrixC;
    //��ʼ��Host
    matrixM = (float*)malloc(M_SIZE * sizeof(float));
    matrixN = (float*)malloc(M_SIZE * sizeof(float));
    matrixC = (float*)malloc(M_SIZE * sizeof(float));
    //��ʼ��GPUm
    hipMalloc((void**)&g_matrixM, sizeof(float) * M_SIZE);
    hipMalloc((void**)&g_matrixN, sizeof(float) * M_SIZE);
    hipMalloc((void**)&g_matrixC, sizeof(float) * M_SIZE);
    //��ֵ
    for (int i = 0; i < M_SIZE; i++)
    {
        matrixM[i] = 7;
        matrixN[i] = 2;
    }
    hipMemcpy(g_matrixM, matrixM, sizeof(float) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_matrixN, matrixN, sizeof(float) * M_SIZE, hipMemcpyHostToDevice);
    dim3 dimGrid(GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE);
    start = clock();

    MatrixMutiplyG << < dimGrid, dimBlock >> > (g_matrixM, g_matrixN, g_matrixC);
    hipDeviceSynchronize();
    finish = clock();
    hipMemcpy(matrixC, g_matrixC, sizeof(float) * M_SIZE, hipMemcpyDeviceToHost);
    printf("GPU_Time:%fs\n", (double)(finish - start) / CLOCKS_PER_SEC);
    printf("vlaue:%f", matrixC[0]);
    int num = 0; 
    //��֤�Ƿ������� 1024*1024=1048576
    for (int i = 0; i < M_SIZE; i++)
    {
        if (matrixC[i] == 14336)
            num++;
    }
    printf("\n%d", num);
    free(matrixM);
    free(matrixN);
    free(matrixC);
    hipFree(g_matrixM);
    hipFree(g_matrixN);
    hipFree(g_matrixC);
    return 0;
}