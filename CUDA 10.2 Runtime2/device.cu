
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include <iostream>
//
//int main()
//{
//    int deviceCount;
//    cudaGetDeviceCount(&deviceCount);
//    for (int i = 0; i < deviceCount; i++)
//    {
//        cudaDeviceProp devProp;
//        cudaGetDeviceProperties(&devProp, i);
//        std::cout << "ʹ��GPU device " << i << ": " << devProp.name << std::endl;
//        std::cout << "�豸ȫ���ڴ������� " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
//        std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
//        std::cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
//        std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
//        std::cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << devProp.regsPerBlock << std::endl;
//        std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
//        std::cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
//        std::cout << "�豸�϶ദ������������ " << devProp.multiProcessorCount << std::endl;
//        std::cout << "======================================================" << std::endl;
//
//    }
//    return 0;
//}