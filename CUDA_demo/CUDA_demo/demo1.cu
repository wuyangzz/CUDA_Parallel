#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include <time.h>

#define MAX_SIZE 1024
#define M_SIZE MAX_SIZE * MAX_SIZE 
#define GRID_SIZE  32
#define BLOCK_SIZE  32

__global__ void MatrixMutiplyG(float* matrixM, float* matrixN, float* matrixC) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0, m, n;
    for (int k = 0; k < MAX_SIZE; ++k) {
        m = matrixM[ty * MAX_SIZE + k];
        n = matrixN[k * MAX_SIZE + tx];
        sum += m * n;
    }
    matrixC[ty * MAX_SIZE + tx] = sum;
}

int main()
{   

    clock_t start, finish;
    float *matrixM, * matrixN, *matrixC;
    float *g_matrixM, * g_matrixN, * g_matrixC;
    //��ʼ��Host
    matrixM = (float*)malloc(M_SIZE *sizeof(float));
    matrixN = (float*)malloc(M_SIZE * sizeof(float));
    matrixC = (float*)malloc(M_SIZE * sizeof(float));
    //��ʼ��GPUm
    hipMalloc((void**)&g_matrixM, sizeof(float) * M_SIZE);
    hipMalloc((void**)&g_matrixN, sizeof(float) * M_SIZE);
    hipMalloc((void**)&g_matrixC, sizeof(float) * M_SIZE);
    //��ֵ
    for (int i = 0; i < M_SIZE; i++)
    {
        matrixM[i] = 7;
        matrixN[i] = 2;
    }
    hipMemcpy(g_matrixM, matrixM, sizeof(float) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_matrixN, matrixN, sizeof(float) * M_SIZE, hipMemcpyHostToDevice);
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    start = clock();

    MatrixMutiplyG << < dimGrid, dimBlock >> > (g_matrixM, g_matrixN, g_matrixC);

    finish = clock();
    hipMemcpy(matrixC, g_matrixC, sizeof(float) * M_SIZE, hipMemcpyDeviceToHost);
    printf("GPU_Time:%fs", (double)(finish - start) / CLOCKS_PER_SEC);
    free(matrixM);
    free(matrixN);
    free(matrixC);
    hipFree(g_matrixM);
    hipFree(g_matrixN);
    hipFree(g_matrixC);
    return 0;
}