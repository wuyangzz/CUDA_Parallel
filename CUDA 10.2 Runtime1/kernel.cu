#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define SIZE 1024 
#define GRID_SIZE 32
#define BLOCK_SIZE 16

__global__ void matrixMultiplication(float* matrixM, float* matrixN, float* matrixP, int width) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0, m, n;
    for (int k = 0; k < width; ++k) {
        m = matrixM[ty * width + k];
        n = matrixN[k * width + tx];
        sum += m * n;
    }
    matrixP[ty * width + tx] = sum;
}


// ������������
int main(void) {

    float* h_matrixM, * h_matrixN, * h_matrixP, * d_matrixM, * d_matrixN, * d_matrixP;
    int mem_size = SIZE * SIZE * sizeof(float);

    // �������ڴ����� A��B��C �����Ŀռ�
    h_matrixM = (float*)malloc(mem_size);
    h_matrixN = (float*)malloc(mem_size);
    h_matrixP = (float*)malloc(mem_size);

    // �� GPU �豸���� A��B��C �����Ŀռ�
    hipMalloc((void**)&d_matrixM, mem_size);
    hipMalloc((void**)&d_matrixN, mem_size);
    hipMalloc((void**)&d_matrixP, mem_size);

    // ��ʼ�������ڴ�� A��B ����
    for (int i = 0; i < SIZE * SIZE; ++i) {
        h_matrixM[i] = 3;
        h_matrixN[i] = 2;
    }

    // ���������ڴ�� A��B �����ݵ� GPU �豸�� A��B
    hipMemcpy(d_matrixM, h_matrixM, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixN, h_matrixN, mem_size, hipMemcpyHostToDevice);

    // GPU �ں˺�����ά�Ȳ���
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // ��¼����ʼ���е�ʱ��
    double startTime, endTime;
    startTime = (double)clock();
    // ִ�� GPU �ں˺���
    matrixMultiplication <<< dimGrid, dimBlock >>> (d_matrixM, d_matrixN, d_matrixP, SIZE);
    // ����������л��ѵ�ʱ��
    endTime = (double)clock();
    printf("Time elapsed: %.2f ms\n", endTime - startTime);


    // �� GPU �豸���ƽ������ C �������ڴ�� C
    hipMemcpy(h_matrixP, d_matrixP, mem_size, hipMemcpyDeviceToHost);

    // ������
    // for (int i = 0; i < SIZE * SIZE; ++i) {
    //     printf("%.2f\n", h_matrixP[0]);
    // }

    printf("end\n");

    free(h_matrixM);
    free(h_matrixN);
    free(h_matrixP);
    hipFree(d_matrixM);
    hipFree(d_matrixN);
    hipFree(d_matrixP);
    return 0;
}
