#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define SIZE 1024 
#define GRID_SIZE 32
#define BLOCK_SIZE 16

__global__ void matrixMultiplication(float* matrixM, float* matrixN, float* matrixP, int width) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0, m, n;
    for (int k = 0; k < width; ++k) {
        m = matrixM[ty * width + k];
        n = matrixN[k * width + tx];
        sum += m * n;
    }
    matrixP[ty * width + tx] = sum;
}


// 主机端主函数
int main(void) {

    float* h_matrixM, * h_matrixN, * h_matrixP, * d_matrixM, * d_matrixN, * d_matrixP;
    int mem_size = SIZE * SIZE * sizeof(float);

    // 在主机内存申请 A，B，C 向量的空间
    h_matrixM = (float*)malloc(mem_size);
    h_matrixN = (float*)malloc(mem_size);
    h_matrixP = (float*)malloc(mem_size);

    // 在 GPU 设备申请 A，B，C 向量的空间
    hipMalloc((void**)&d_matrixM, mem_size);
    hipMalloc((void**)&d_matrixN, mem_size);
    hipMalloc((void**)&d_matrixP, mem_size);

    // 初始化主机内存的 A，B 向量
    for (int i = 0; i < SIZE * SIZE; ++i) {
        h_matrixM[i] = 3;
        h_matrixN[i] = 2;
    }

    // 拷贝主机内存的 A，B 的内容到 GPU 设备的 A，B
    hipMemcpy(d_matrixM, h_matrixM, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixN, h_matrixN, mem_size, hipMemcpyHostToDevice);

    // GPU 内核函数的维度参数
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // 记录程序开始运行的时间
    double startTime, endTime;
    startTime = (double)clock();
    // 执行 GPU 内核函数
    matrixMultiplication <<< dimGrid, dimBlock >>> (d_matrixM, d_matrixN, d_matrixP, SIZE);
    // 输出程序运行花费的时间
    endTime = (double)clock();
    printf("Time elapsed: %.2f ms\n", endTime - startTime);


    // 从 GPU 设备复制结果向量 C 到主机内存的 C
    hipMemcpy(h_matrixP, d_matrixP, mem_size, hipMemcpyDeviceToHost);

    // 输出结果
    // for (int i = 0; i < SIZE * SIZE; ++i) {
    //     printf("%.2f\n", h_matrixP[0]);
    // }

    printf("end\n");

    free(h_matrixM);
    free(h_matrixN);
    free(h_matrixP);
    hipFree(d_matrixM);
    hipFree(d_matrixN);
    hipFree(d_matrixP);
    return 0;
}
